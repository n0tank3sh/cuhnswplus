#include "hip/hip_runtime.h"
// Copyright (c) 2020 Jisang Yoon
// All rights reserved.
//
// This source code is licensed under the Apache 2.0 license found in the
// LICENSE file in the root directory of this source tree.
#include <iostream>
#include <algorithm>
#include <random>
#include <cmath>

#include "cuhnswplus.hpp"
#include "cuda_build_kernels.cuh"

namespace cuhnswplus {

void CuHNSW::GetDeviceInfo() {
  CHECK_CUDA(hipGetDevice(&devId_));
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDeviceProperties(&prop, devId_));
  mp_cnt_ = prop.multiProcessorCount;
  major_ = prop.major;
  minor_ = prop.minor;
  cores_ = -1;
}
inline int GetCores(int major, int minor, int mp_cnt) {      
  int cores = -1;
  switch (major){
    case 2: // Fermi
      if (minor == 1) 
        cores = mp_cnt * 48;
      else 
        cores = mp_cnt * 32;
      break;
    case 3: // Kepler
      cores = mp_cnt * 192;
      break;
    case 5: // Maxwell
      cores = mp_cnt * 128;
      break;
    case 6: // Pascal
      if (minor == 1 or minor == 2) 
        cores = mp_cnt * 128;
      else if (minor == 0) 
        cores = mp_cnt * 64;
      else 
      //  DEBUG0("Unknown device type");
      break;
    case 7: // Volta and Turing
      if (minor == 0 or minor == 5) 
        cores = mp_cnt * 64;
      else 
     //   DEBUG0("Unknown device type");
      break;
    case 8: // Ampere
      if (minor == 0) 
        cores = mp_cnt * 64;
      else if (minor == 6) 
        cores = mp_cnt * 128;
      else 
    //    DEBUG0("Unknown device type");
      break;
    default:
   //   DEBUG0("Unknown device type"); 
      break;
  }
  if (cores == -1) cores = mp_cnt * 128;
  return cores;
}
void CuHNSW::SelectGPU(int gpu_id) {
  if(gpus.size() == 0) {
    int dcount = 0;
    CHECK_CUDA(hipGetDeviceCount(&dcount));
    int device;
    gpus.resize(dcount);
    hipDeviceProp_t prop;
    for(int i = 0; i < dcount; i++) {
      CHECK_CUDA(hipSetDevice(i));
      CHECK_CUDA(hipGetDevice(&device));
      CHECK_CUDA(hipGetDeviceProperties(&prop, device));
      int mp_cnt = prop.multiProcessorCount;
      int major = prop.major;
      int minor = prop.minor; 
      int block_cnt = opt_["hyper_threads"].number_value() * (GetCores(major, minor, mp_cnt) / block_dim_);
      gpus[i] = block_cnt;
    }
  }
  assert(gpu_id < gpus.size());
  CHECK_CUDA(hipSetDevice(gpu_id));
  block_cnt_ = gpus[gpu_id];
}

void CuHNSW::SetDims(int dims) {
  num_dims_ = dims;
}


void CuHNSW::BuildGraph() {
  visited_ = new bool[batch_size_ * num_data_];
  //for (int level = max_level_; level >= 0; --level) {
  //  DEBUG("build graph of level: {}", level);
  //  BuildLevelGraph(level);
  //}
  BuildLevelGraph(0);
}

void CuHNSW::BuildLevelGraph(int level) {
  
  int max_size = level_graphs_[0].GetNodes().size();
  thrust::device_vector<int> device_graph(max_m0_ * max_size);
  thrust::device_vector<float> device_distances(max_m0_ * max_size);
  thrust::device_vector<int> device_deg(max_size);
  thrust::device_vector<int> device_nodes(max_size);
  thrust::device_vector<int> device_visited_table(visited_table_size_ * block_cnt_, -1);
  thrust::device_vector<int> device_visited_list(visited_list_size_ * block_cnt_);
  thrust::device_vector<int> device_mutex(max_size, 0);
  thrust::device_vector<int64_t> device_acc_visited_cnt(block_cnt_, 0);
  thrust::device_vector<Neighbor> device_neighbors(ef_construction_ * block_cnt_);
  thrust::device_vector<int> device_cand_nodes(ef_construction_ * block_cnt_);
  thrust::device_vector<cuda_scalar> device_cand_distances(ef_construction_ * block_cnt_);
  thrust::device_vector<int> device_backup_neighbors(max_m0_ * block_cnt_);
  thrust::device_vector<cuda_scalar> device_backup_distances(max_m0_ * block_cnt_);
  thrust::device_vector<bool> device_went_through_heuristic(max_size, false);

  for(int l = max_level_; l >= 0; l--) {
    std::set<int> upper_nodes;
    std::vector<int> new_nodes;
    LevelGraph& graph = level_graphs_[l];
    const std::vector<int>& nodes = graph.GetNodes();
    int size = nodes.size();  
    int max_m = l > 0? max_m_: max_m0_;

    std::vector<int> graph_vec(size * max_m, 0);
    std::vector<int> deg(size, 0);
    if (l < max_level_) {
      LevelGraph& upper_graph = level_graphs_[l + 1];
      upper_graph.LoadGraphVec(graph_vec, deg, max_m);

    }

    for (auto& node: graph.GetNodes()) {
      if (upper_nodes.count(node)) continue;
      new_nodes.push_back(node);
    }
  
    // initialize entries
    std::vector<int> entries(new_nodes.size(), enter_point_);

    GetEntryPoints(new_nodes, entries, l, false);
    for (int i = 0; i < new_nodes.size(); ++i) {
      int srcid = graph.GetNodeId(new_nodes[i]);
      int dstid = graph.GetNodeId(entries[i]);
      graph_vec[max_m * srcid] = dstid;
      deg[srcid] = 1;
    }

    thrust::copy(graph_vec.begin(), graph_vec.end(), device_graph.begin());
    thrust::copy(deg.begin(), deg.end(), device_deg.begin());
    thrust::copy(nodes.begin(), nodes.end(), device_nodes.begin());

    BuildLevelGraphKernel<<<block_cnt_, block_dim_>>>(
      thrust::raw_pointer_cast(device_data_.data()),
      thrust::raw_pointer_cast(device_nodes.data()),
      num_dims_, size, max_m, dist_type_, save_remains_,
      ef_construction_,
      thrust::raw_pointer_cast(device_graph.data()),
      thrust::raw_pointer_cast(device_distances.data()),
      thrust::raw_pointer_cast(device_deg.data()),
      thrust::raw_pointer_cast(device_visited_table.data()),
      thrust::raw_pointer_cast(device_visited_list.data()),
      visited_table_size_, visited_list_size_,
      thrust::raw_pointer_cast(device_mutex.data()),
      thrust::raw_pointer_cast(device_acc_visited_cnt.data()),
      reverse_cand_,
      thrust::raw_pointer_cast(device_neighbors.data()),
      thrust::raw_pointer_cast(device_cand_nodes.data()),
      thrust::raw_pointer_cast(device_cand_distances.data()),
      heuristic_coef_,
      thrust::raw_pointer_cast(device_backup_neighbors.data()),
      thrust::raw_pointer_cast(device_backup_distances.data()),
      thrust::raw_pointer_cast(device_went_through_heuristic.data())
      );
    CHECK_CUDA(hipDeviceSynchronize());
    thrust::copy(device_deg.begin(), device_deg.begin() + deg.size(), deg.begin());
    thrust::copy(device_graph.begin(), device_graph.begin() + graph_vec.size(), graph_vec.begin());
    std::vector<float> distances(max_m * size);
    thrust::copy(device_distances.begin(), device_distances.begin() + distances.size(), distances.begin());

    std::vector<int64_t> acc_visited_cnt(block_cnt_);
    thrust::copy(device_acc_visited_cnt.begin(), device_acc_visited_cnt.end(), acc_visited_cnt.begin());
    CHECK_CUDA(hipDeviceSynchronize());
    int64_t full_visited_cnt = std::accumulate(acc_visited_cnt.begin(), acc_visited_cnt.end(), 0LL);
    DEBUG("full number of visited nodes: {}", full_visited_cnt);

    graph.UnLoadGraphVec(graph_vec, deg, distances, max_m);
  }
}

void CuHNSW::AddPoint(const float* qdata, int level, int label) {  
  #ifdef HALF_PRECISION
    // DEBUG0("fp16")
    std::vector<cuda_scalar> hdata(num_dims_);
    for (int i = 0; i <  num_dims_; ++i) {
      hdata[i] = conversion(qdata[i]);
      // DEBUG("hdata i: {}, scalar: {}", i, out_scalar(hdata[i]));
    }
    device_data_.insert(device_data_.end(), hdata.begin(), hdata.end());
  #else
    // DEBUG0("fp32")
    device_data_.insert(device_data_.end(), qdata, qdata + num_dims_);
  #endif
  data_.insert(data_.end(), qdata, qdata + num_dims_);
  if(level == -1) {
    level = (int)(-std::log(std::uniform_real_distribution(0.0, 1.0)(level_generator)) * level_mult_);
  }
  levels_.resize(num_data_ + 1);
  levels_[num_data_] = level;
  if(level > max_level_) {
    enter_point_ = num_data_;
    max_level_ = level;
    level_graphs_.resize(level + 1, LevelGraph(max_elements_));
  } 
  for(int i = 0; i <= level; i++) {
    level_graphs_[i].AddNode(num_data_);
  }
  if(labelled_) {
    labels_.push_back(label);
  }
  num_data_++;
}
void CuHNSW::AddPoints(const float* qdata, int* levels, int* labels, int num_points) {
  for(int i = 0; i < num_points; i++) {
    AddPoint(qdata + i * num_dims_, levels[i], labels[i]);
  }
}

} // namespace cuhnswplus
